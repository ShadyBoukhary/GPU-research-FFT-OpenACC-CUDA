#include "hip/hip_runtime.h"
/*+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-++-+-+-+-+-+-+-+-+-+-+-+-+-+-+-++-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
*   Shady Boukhary
*   Midwestern State University
*   Research Week 5 - CUDA - Fast Fourier Transform: Cooley Tukey
*   March 23rd, 2018
*
*
*   Computes a radix-2 fast fourier transform using an iterative implementation of
*	the Cooley-Tukey Algorithm. This code uses CUDA 
*
*	To compile: nvcc -o fft FFT_CudaG.cu
+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-++-+-+-+-+-+-+-+-+-+-+-+-+-+-+-++-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+*/

#include <stdio.h>
#include <stdlib.h>      
#include <complex.h>
#include <hip/hip_complex.h>    
#include <math.h>
#include "timer.h"
#include <hip/hip_runtime.h>

#define PI 3.14159265
#define SIZE 1048576
#define TILE_SIZE 1024

//double _Complex * computeFFT(double _Complex *, int);

__device__ 
hipDoubleComplex eIThetta(int, int, int, int);
void printFFT(const hipDoubleComplex *, int);


__global__ void FFTKernel(hipDoubleComplex * numbers, hipDoubleComplex * X, int N) {
    int tx = threadIdx.x + TILE_SIZE * blockIdx.x;
    
    __shared__ hipDoubleComplex numbers_SM[TILE_SIZE];

    // copy 1 tile from global memory into shared memory
    // i: phase
    hipDoubleComplex sumEven = make_hipDoubleComplex(0, 0); 
	hipDoubleComplex sumOdd = make_hipDoubleComplex(0, 0);
    for (int i = 0; i < SIZE / TILE_SIZE; i++) {
        numbers_SM[threadIdx.x] = numbers[i * TILE_SIZE + threadIdx.x];
        // make sure all threads copied their part
        __syncthreads();

        for (int n = 0; n < (TILE_SIZE / 2); n++) {
            // compute the even part

		hipDoubleComplex comp = numbers_SM[2 * n];
		hipDoubleComplex eThetta = eIThetta(tx, N, (n + (i * (TILE_SIZE / 2))), 0);
		hipDoubleComplex resultEven = hipCmul(comp, eThetta);
		sumEven = hipCadd(resultEven, sumEven);

		// compute the odd part

        hipDoubleComplex compOdd = numbers_SM[2 * n + 1];

        hipDoubleComplex eThettaOdd = eIThetta(tx, N, (n + (i * (TILE_SIZE / 2))), 1);

		hipDoubleComplex resultOdd = hipCmul(compOdd, eThettaOdd);
        sumOdd = hipCadd(resultOdd, sumOdd);
        
        }
        // make sure all threads computed current phase
        __syncthreads();  
    }

    X[tx] = hipCadd(sumEven, sumOdd);

}

int main()
{
	double start, stop, elapsed;

	hipDoubleComplex * signals = (hipDoubleComplex*)malloc(SIZE * sizeof(hipDoubleComplex));
    hipDoubleComplex * fft = (hipDoubleComplex*)malloc(SIZE * sizeof(hipDoubleComplex));
    hipDoubleComplex * signalsD;
    hipDoubleComplex * fftD;
	
	double size = SIZE * sizeof(hipDoubleComplex);

	for (int x = 0; x < SIZE; x++) {
		signals[x] = make_hipDoubleComplex(x, SIZE - x); //x + (SIZE - x) * I;
	}
	hipMalloc((void **)&signalsD, size);
	hipMalloc((void **)&fftD, size);

	hipMemcpy(signalsD, signals, size, hipMemcpyHostToDevice);

	dim3 dimGrid0(SIZE / 1024, 1, 1);
	dim3 dimBlock0(1024, 1, 1);
	GET_TIME(start);
	FFTKernel<<<dimGrid0, dimBlock0>>>(signalsD, fftD, SIZE);
	//fft = computeFFT(signals, SIZE);
	hipDeviceSynchronize();
	GET_TIME(stop);
	elapsed = stop - start;

	hipMemcpy(fft, fftD, size, hipMemcpyDeviceToHost);
	printFFT(fft, SIZE);
	printf("Code to be timed took %e seconds.\n", elapsed);
	//getchar();
	hipFree(signalsD);
	hipFree(fftD);

}


/*+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-
*   eIThetta()
*   @param: int, int, int, int
*   @return: double _Complex
*   Description: computes the spin of the signal around a circle at its frequency
+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-*/
__device__
hipDoubleComplex eIThetta(int k, int N, int n, int offset) {
	// compute real part
	double realPart = cos((2 * PI * (2 * n + offset) * k) / N);

	// compute imaginary part
	double imaginaryPart = (-1) * sin((2 * PI * (2 * n + offset) * k) / N);

	// create a _Complex number out of them and return it
	hipDoubleComplex result = make_hipDoubleComplex(realPart, imaginaryPart);//realPart + imaginaryPart * I;
	return result;
}

/*+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-
*   printFFT()
*   @param: double _Complex, int
*   @return: none
*   Description: prints the FFT (components of the signal etc..)
+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-*/
void printFFT(const hipDoubleComplex * fft, int N) {
	//for (int i = 0; i < N; i++) {
		//printf("X(%i) = %f + %fi\n", i, creal(fft[i]), cimag(fft[i]));
	//}
	printf("X(1) = %f + %fi\nX(N-1) = %f + %fi\n", hipCreal(fft[1]), hipCimag(fft[1]), hipCreal(fft[N - 1]), hipCimag(fft[N - 1]));
}


